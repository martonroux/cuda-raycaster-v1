/*
** RAYCASTING LIBRARY
** cudaHelpers.cu
** Created by marton on 29/06/24.
*/

#include "cudaHelpers.h"
#include <iostream>

void checkCudaError(hipError_t result, const char *func)
{
    if (result != hipSuccess) {
        std::cerr << "CUDA error in " << func << ": " << hipGetErrorString(result) << std::endl;
        exit(result);
    }
}
