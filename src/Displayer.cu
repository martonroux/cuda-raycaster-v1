/*
** RAYCASTING LIBRARY
** Displayer.cu
** Created by marton on 29/06/24.
*/

#include "Displayer.hpp"

namespace rcr {

    void Displayer::tempCreateImage(rcr::matrix3<rcr::hitPos> image) {
        keyboard_.resetPresses();
        mouse_.resetPresses();

        for (int i = 0; i < width_; i++) {
            for (int j = 0; j < height_; j++) {
                img_.at<cv::Vec3b>(j, i)[0] = image(j, i, 0, nullptr).hit ? 255 : 0;
                if (shapes_.size() > 1)
                    img_.at<cv::Vec3b>(j, i)[1] = image(j, i, 1, nullptr).hit ? 255 : 0;
                if (shapes_.size() > 2)
                    img_.at<cv::Vec3b>(j, i)[2] = image(j, i, 2, nullptr).hit ? 255 : 0;
            }
        }
        cv::imshow("Raycaster", img_);
        int key = cv::waitKey(static_cast<int>(1.f / static_cast<float>(fps_) * 1000));

        keyboard_.setKeyPressed(static_cast<Keys>(key), true);
    }

    std::pair<int, int> Displayer::getNumThreadsBlocks() const
    {
        int numBlocks = (static_cast<int>(shapes_.size() * width_ * height_) + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;

        return {numBlocks, NUM_THREADS_PER_BLOCK};
    }

    // matrix3<rcr::hitPos> *Displayer::createHitMatrix() const
    // {
    //     rcr::hitPos *d_values;
    //
    //     hipMalloc((void**)&d_values, sizeof(rcr::hitPos) * height_ * width_ * shapes_.size());
    //
    //     rcr::matrix3<rcr::hitPos> h_image{height_, width_, shapes_.size(), d_values};
    //     rcr::matrix3<rcr::hitPos> *d_image;
    //
    //     hipMalloc((void **) &d_image, sizeof(rcr::matrix3<rcr::hitPos>));
    //     hipMemcpy(d_image, &h_image, sizeof(rcr::matrix3<rcr::hitPos>), hipMemcpyHostToDevice);
    //
    //     return d_image;
    // }
    //
    // matrix3<rcr::hitPos> Displayer::retrieveDeviceMatrix(matrix3<rcr::hitPos> *d_matrix, size_t row, size_t col,
    //                                                      size_t dep)
    // {
    //     auto *h_matrix = (matrix3<rcr::hitPos>*)malloc(sizeof(matrix3<rcr::hitPos>));
    //     auto *h_values = (rcr::hitPos*)malloc(sizeof(rcr::hitPos) * row * col * dep);
    //
    //     hipMemcpy(h_matrix, d_matrix, sizeof(rcr::matrix3<rcr::hitPos>), hipMemcpyDeviceToHost);
    //     hipMemcpy(h_values, h_matrix->getValues(), sizeof(rcr::hitPos) * row * col * dep, hipMemcpyDeviceToHost);
    //
    //     return matrix3{row, col, dep, h_values};
    // }

    Triangle *Displayer::createTriangleArray() const
    {
        auto *h_triangles = (Triangle *)malloc(sizeof(Triangle) * shapes_.size());
        Triangle *d_triangles;

        for (int i = 0; i < static_cast<int>(shapes_.size()); i++) {
            memcpy(&h_triangles[i], &shapes_[i], sizeof(Triangle));
        }

        checkCudaError(hipMalloc((void**)&d_triangles, sizeof(rcr::Triangle) * shapes_.size()), "hipMalloc d_triangle");
        checkCudaError(hipMemcpy(d_triangles, h_triangles, sizeof(rcr::Triangle) * shapes_.size(), hipMemcpyHostToDevice), "hipMemcpy d_triangle");

        free(h_triangles);
        return d_triangles;
    }

    hitPos *Displayer::getDeviceHits() const {
        hitPos *d_hits;

        hipMalloc((void**)&d_hits, sizeof(hitPos) * height_ * width_ * shapes_.size());
        return d_hits;
    }

    hitPos *Displayer::moveHitsToHost(hitPos *d_hits) const {
        auto *h_hits = (hitPos*)malloc(sizeof(hitPos) * height_ * width_ * shapes_.size());

        hipMemcpy(h_hits, d_hits, sizeof(hitPos) * height_ * width_ * shapes_.size(), hipMemcpyDeviceToHost);
        return h_hits;
    }

    Displayer::Displayer(size_t width, size_t height, size_t fps, rendererData data) : height_(height), width_(width), fps_(fps), screen_(data), img_(height, width, CV_8UC3, cv::Scalar(0, 0, 0))
    {
        cv::namedWindow("Raycaster");
        cv::setMouseCallback("Raycaster", rcr::onMouseCallback, &mouse_);
    }

    Displayer::~Displayer()
    {
        cv::destroyWindow("Raycaster");
    }

    void Displayer::addShape(Triangle triangle)
    {
        shapes_.push_back(triangle);
    }

    void Displayer::render()
    {
        hitPos *d_hits = getDeviceHits();
        hitPos *h_hits;
        Triangle *d_triangles = createTriangleArray();
        CudaError *d_error = CudaError::createDeviceCudaError();
        std::pair<int, int> dimensions = getNumThreadsBlocks();

        kernelRender<<<dimensions.first, dimensions.second>>>(d_hits, height_, width_, shapes_.size(), d_triangles, screen_, d_error);

        h_hits = moveHitsToHost(d_hits);

        // tempCreateImage(matrix3{height_, width_, shapes_.size(), h_hits});

        free(h_hits);

        hipFree(d_hits);
        hipFree(d_triangles);
        hipFree(d_error);
    }

    void Displayer::clear() {
        img_ = cv::Mat(static_cast<int>(height_), static_cast<int>(width_), CV_8UC3, cv::Scalar(0, 0, 0));
    }

    void Displayer::clear(rgb backgroundColor) {
        img_ = cv::Mat(static_cast<int>(height_), static_cast<int>(width_), CV_8UC3, cv::Scalar(backgroundColor.b, backgroundColor.g, backgroundColor.r));
    }

    Keyboard Displayer::getKeyboardFrame() const {
        return keyboard_;
    }

    Mouse Displayer::getMouseFrame() const {
        return mouse_;
    }
} // rcr
