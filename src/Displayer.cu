/*
** RAYCASTING LIBRARY
** Displayer.cu
** Created by marton on 29/06/24.
*/

#include "Displayer.hpp"

namespace rcr {

    void tempCreateImage(rcr::matrix3<rcr::hitPos> image, size_t height, size_t width, size_t nbTriangles) {
        cv::Mat temp(height, width, CV_8UC3, cv::Scalar(0, 0, 0));

        for (int i = 0; i < width; i++) {
            for (int j = 0; j < height; j++) {
                temp.at<cv::Vec3b>(j, i)[0] = image(j, i, 0, nullptr).hit ? 255 : 0;
                if (nbTriangles > 1)
                    temp.at<cv::Vec3b>(j, i)[1] = image(j, i, 1, nullptr).hit ? 255 : 0;
                if (nbTriangles > 2)
                    temp.at<cv::Vec3b>(j, i)[2] = image(j, i, 2, nullptr).hit ? 255 : 0;
            }
        }
        cv::imshow("Raycaster", temp);
        cv::waitKey(0);
    }

    std::pair<int, int> Displayer::getNumThreadsBlocks(unsigned int numThreadsPerBlock) const
    {
        int numBlocks = (shapes_.size() * width_ * height_ + numThreadsPerBlock - 1) / numThreadsPerBlock;

        return {numBlocks, numThreadsPerBlock};
    }

    matrix3<rcr::hitPos> *Displayer::createHitMatrix() const
    {
        rcr::matrix3<rcr::hitPos> h_image{height_, width_, shapes_.size()};
        rcr::matrix3<rcr::hitPos> *d_image;

        h_image.moveToDevice();

        hipMalloc((void **) &d_image, sizeof(rcr::matrix3<rcr::hitPos>));
        hipMemcpy(d_image, &h_image, sizeof(rcr::matrix3<rcr::hitPos>), hipMemcpyHostToDevice);

        return d_image;
    }

    matrix3<rcr::hitPos> Displayer::retrieveDeviceMatrix(matrix3<rcr::hitPos> *d_matrix, size_t row, size_t col,
                                                         size_t dep)
    {
        matrix3<rcr::hitPos> h_matrix{row, col, dep};

        hipMemcpy(&h_matrix, d_matrix, sizeof(rcr::matrix3<rcr::hitPos>), hipMemcpyDeviceToHost);
        h_matrix.moveToHost();
        return h_matrix;
    }

    Triangle *Displayer::createTriangleArray() const
    {
        Triangle *h_triangles = (Triangle *)malloc(sizeof(Triangle) * shapes_.size());
        Triangle *d_triangles;

        for (int i = 0; i < static_cast<int>(shapes_.size()); i++) {
            memcpy(&h_triangles[i], &shapes_[i], sizeof(Triangle));
        }

        checkCudaError(hipMalloc((void**)&d_triangles, sizeof(rcr::Triangle) * shapes_.size()), "hipMalloc d_triangle");
        checkCudaError(hipMemcpy(d_triangles, h_triangles, sizeof(rcr::Triangle) * shapes_.size(), hipMemcpyHostToDevice), "hipMemcpy d_triangle");

        return d_triangles;
    }

    Displayer::Displayer(size_t width, size_t height, size_t fps, rendererData data) : height_(height), width_(width), screen_(data), img_(height, width, CV_8UC3, cv::Scalar(0, 0, 0)) {
    }

    void Displayer::addShape(Triangle triangle) {
        shapes_.push_back(triangle);
    }

    void Displayer::render() {
        matrix3<rcr::hitPos> *d_hits = createHitMatrix();
        Triangle *d_triangles = createTriangleArray();
        CudaError *d_error = rcr::CudaError::createDeviceCudaError();
        std::pair<int, int> dimensions = getNumThreadsBlocks(512);

        kernelRender<<<dimensions.first, dimensions.second>>>(d_hits, height_, width_, shapes_.size(), d_triangles, screen_, d_error);

        matrix3<rcr::hitPos> h_hits = retrieveDeviceMatrix(d_hits, height_, width_, shapes_.size());
        tempCreateImage(h_hits, height_, width_, shapes_.size());
    }

    void Displayer::clear() {
        img_ = cv::Mat(height_, width_, CV_8UC3, cv::Scalar(0, 0, 0));
    }

    void Displayer::clear(rgb backgroundColor) {
        img_ = cv::Mat(height_, width_, CV_8UC3, cv::Scalar(backgroundColor.b, backgroundColor.g, backgroundColor.r));
    }
} // rcr
